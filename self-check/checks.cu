#include "checks.cuh"
#include <stdio.h>

#define CUDA_ERROR_CHECK(err, msg) {    \
    if ((err) != hipSuccess) {         \
        fprintf(stderr, "%s: cuda error=%d - %s\n", (msg), (int)(err), hipGetErrorString(err));  \
        return -1;                      \
    }                                   \
}

int get_devices_count() {
    int result;
    auto err = hipGetDeviceCount(&result);
    CUDA_ERROR_CHECK(err, "Cannot get device count");
    return result;
}

int get_device_name(int device, char** result) {
    if (result == nullptr) return -2;
    hipDeviceProp_t prop;
    auto err = hipGetDeviceProperties(&prop, device);
    CUDA_ERROR_CHECK(err, "Cannot get device properties");
    *result = prop.name;
    return 0;
}

int device_malloc(int device, void** result) {
    if (result == nullptr) return -2;
    auto err = hipSetDevice(device);
    CUDA_ERROR_CHECK(err, "Cannot set active device");
    void* mem = nullptr;
    err = hipMalloc(&mem, 1024);
    CUDA_ERROR_CHECK(err, "Cannot allocate memory");
    *result = mem;
    return 0;
}

int device_free(int device, void* ptr) {
    if (ptr == nullptr) return -2;
    auto err = hipSetDevice(device);
    CUDA_ERROR_CHECK(err, "Cannot set active device");
    err = hipFree(ptr);
    CUDA_ERROR_CHECK(err, "Cannot free memory");
    return 0;
}
